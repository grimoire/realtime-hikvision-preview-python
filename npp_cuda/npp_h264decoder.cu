#include "npp_h264decoder.cuh"
#include <iostream>
#include <npp.h>
#include <nppi.h>
#include <npps.h>

void decodeH264(const unsigned char *input, unsigned char *output, int width,
                int height,int device_id) {

  int curDev = -1;
  hipGetDevice(&curDev);
  if(curDev!=device_id){
    hipSetDevice(device_id);
  }

  Npp8u *pNppInput;
  int nppInputStep;
  pNppInput = nppiMalloc_8u_C1(width, height/2*3, &nppInputStep);
  hipMemcpy(pNppInput, input, sizeof(Npp8u) * width * height/2*3,
             hipMemcpyHostToDevice);

  Npp8u *pNppInput_V_half = pNppInput + width * height;

  Npp8u *pNppInput_U_half = pNppInput_V_half + width * height /4;

  Npp8u *pNppInputArray[3] = {pNppInput, pNppInput_U_half, pNppInput_V_half};
  int nppInputSteps[3] = {width, width/2, width/2};

  Npp8u *pNppOutput;
  int nppOutputStep;
  pNppOutput = nppiMalloc_8u_C3(width, height, &nppOutputStep);

  NppiSize nppSize;
  nppSize.width = width;
  nppSize.height = height;
  nppiYUV420ToBGR_8u_P3C3R(pNppInputArray, nppInputSteps, pNppOutput, width * 3,
                           nppSize);

  hipMemcpy(output, pNppOutput, sizeof(unsigned char) * width * height * 3,
             hipMemcpyDeviceToHost);

  hipFree(pNppInput);
  hipFree(pNppOutput);

  if(curDev!=device_id){
    hipSetDevice(curDev);
  }
}